#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>


extern "C" void cublas_matrix_multiply(float* h_A, float* h_B, float* h_C, int m, int n, int k) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m * k * sizeof(float));
    hipMalloc((void**)&d_B, k * n * sizeof(float));
    hipMalloc((void**)&d_C, m * n * sizeof(float));

    hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, 
                &alpha, d_B, n, d_A, k, &beta, d_C, n);

    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipblasDestroy(handle);
}
