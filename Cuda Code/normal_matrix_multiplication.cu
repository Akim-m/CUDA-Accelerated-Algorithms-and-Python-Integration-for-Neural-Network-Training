#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel for basic matrix multiplication
__global__ void matrixMulKernel(float* A, float* B, float* C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n) {
        float value = 0;
        for (int i = 0; i < k; ++i) {
            value += A[row * k + i] * B[i * n + col];
        }
        C[row * n + col] = value;
    }
}

// Host function to perform matrix multiplication on GPU
extern "C" void normal_matrix_multiply(float* h_A, float* h_B, float* h_C, int m, int n, int k) {
    // Allocate memory on GPU
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m * k * sizeof(float));
    hipMalloc((void**)&d_B, k * n * sizeof(float));
    hipMalloc((void**)&d_C, m * n * sizeof(float));

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice);

    // Define block size and grid size
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((n + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                       (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch matrix multiplication kernel
    matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, m, n, k);

    // Copy result matrix C back to host
    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
